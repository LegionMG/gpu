
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>


using namespace std;

#define BLOCK_SIZE 256 
#define SEED 1337
#define ARR_LENGTH 5000

__global__ void sum_cud(float * in, float * out, int len) {
    __shared__ float sum[2*BLOCK_SIZE];
    unsigned int th_num = threadIdx.x;
    unsigned int pointer = blockIdx.x * blockDim.x;
    if (pointer + th_num < len)
       sum[th_num] = in[pointer + th_num];
    else
       sum[th_num] = 0;
    for (unsigned int stride = blockDim.x/2; stride >= 1; stride >>= 1) {
       if (th_num < stride)
          sum[th_num] += sum[th_num+stride];
       __syncthreads();
    }
    if (th_num == 0)
       out[blockIdx.x] = sum[0];
}



float randInRange(float min, float max)
{
  return min + (float) (rand() / (double) (RAND_MAX + 1) * (max - min + 1));
}

int main(int argc, char ** argv) {
    srand(SEED);
    float * input;
    float * output;
    float * d_input;
    float * d_output;
    int lenInput = ARR_LENGTH; 
    int lenOutput; 

    input = (float*) malloc(lenInput * sizeof(float));

    for (int i = 0; i < lenInput; ++i)
    {
        input[i] = randInRange(0.0, 10.0);
    }

    clock_t Time;
    Time = clock(); 

    float sum = 0.0;
    for (int i = 0; i < lenInput; i++) {
        sum += input[i];
    }

    cout << "CPU result: " << sum << endl;

    Time = clock() - Time;
    float Time_ = (float) Time / CLOCKS_PER_SEC;
    cout << "CPU time is: " <<  Time_ << endl; 




    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_; 
    hipEventRecord(start, 0);  
    do
    {   

        lenOutput = lenInput / (BLOCK_SIZE);
        if (lenInput % (BLOCK_SIZE)) {
            lenOutput++;
        }
        output = (float*) malloc(lenOutput * sizeof(float));

        hipMalloc(&d_input, sizeof(float) * lenInput);
        hipMalloc(&d_output, sizeof(float) * lenInput);


        hipMemcpy(d_input, input, sizeof(float) * lenInput, hipMemcpyHostToDevice);

        dim3 dimGrid(lenOutput);
        dim3 dimBlock(BLOCK_SIZE);


        sum_cud<<<dimGrid, dimBlock>>>(d_input, d_output, lenInput);

        hipMemcpy(output, d_output, sizeof(float) * lenOutput, hipMemcpyDeviceToHost);
        input = output;
        lenInput = lenOutput; 

    } while (lenOutput > 1);
   
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_, start, stop);
    printf("GPU ime is: %f\n", time_/1000); 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();

    cout << "Cuda result: " << output[0] << endl;

    hipFree(d_input);
    hipFree(d_output);


    free(input);
    free(output);


    return 0;
}